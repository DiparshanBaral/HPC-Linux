#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Decryption logic
__device__ char* CudaDecrypt(char* encryptedPassword) {
    char* decryptedPassword = (char*)malloc(sizeof(char) * 5);

    decryptedPassword[0] = encryptedPassword[0] - 2;
    decryptedPassword[1] = encryptedPassword[3] - 3;
    decryptedPassword[2] = encryptedPassword[6] - 2;
    decryptedPassword[3] = encryptedPassword[8] - 4;
    decryptedPassword[4] = '\0';

    return decryptedPassword;
}

// Kernel function for decryption
__global__ void decryptPassword(char* inputEncryptedPassword, char* decryptedPassword) {
    char* result = CudaDecrypt(inputEncryptedPassword);
    for (int i = 0; i < 5; i++) {
        decryptedPassword[i] = result[i];
    }
}

// Main function
int main() {
    char inputEncryptedPassword[11];
    printf("Enter the 10-character encrypted password to decrypt: ");
    scanf("%10s", inputEncryptedPassword);

    char* gpuInputEncryptedPassword;
    char* gpuDecryptedPassword;
    char decryptedPassword[5];

   
    hipMalloc((void**)&gpuInputEncryptedPassword, sizeof(char) * 11);
    hipMalloc((void**)&gpuDecryptedPassword, sizeof(char) * 5);

    
    hipMemcpy(gpuInputEncryptedPassword, inputEncryptedPassword, sizeof(char) * 11, hipMemcpyHostToDevice);

    
    decryptPassword<<<1, 1>>>(gpuInputEncryptedPassword, gpuDecryptedPassword);

    hipMemcpy(decryptedPassword, gpuDecryptedPassword, sizeof(char) * 5, hipMemcpyDeviceToHost);

    
    printf("Decrypted password: %s\n", decryptedPassword);

   
    hipFree(gpuInputEncryptedPassword);
    hipFree(gpuDecryptedPassword);

    return 0;
}
