#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Device-side encryption logic
__device__ void CudaEncrypt(const char* rawPassword, char* encryptedPassword) {
    encryptedPassword[0] = rawPassword[0] + 2;
    encryptedPassword[1] = rawPassword[0] - 2;
    encryptedPassword[2] = rawPassword[0] + 1;
    encryptedPassword[3] = rawPassword[1] + 3;
    encryptedPassword[4] = rawPassword[1] - 3;
    encryptedPassword[5] = rawPassword[1] - 1;
    encryptedPassword[6] = rawPassword[2] + 2;
    encryptedPassword[7] = rawPassword[2] - 2;
    encryptedPassword[8] = rawPassword[3] + 4;
    encryptedPassword[9] = rawPassword[3] - 4;
    encryptedPassword[10] = '\0';
}

// Device-side decryption logic
__device__ void CudaDecrypt(const char* encryptedPassword, char* decryptedPassword) {
    decryptedPassword[0] = encryptedPassword[0] - 2; // Reverse +2
    decryptedPassword[1] = encryptedPassword[3] - 3; // Reverse +3
    decryptedPassword[2] = encryptedPassword[6] - 2; // Reverse +2
    decryptedPassword[3] = encryptedPassword[8] - 4; // Reverse +4
    decryptedPassword[4] = '\0'; // Null-terminate the string
}

// Kernel function for encryption
__global__ void encryptKernel(const char* rawPassword, char* encryptedPassword) {
    CudaEncrypt(rawPassword, encryptedPassword);
    printf("Raw Password: %s -> Encrypted Password: %s\n", rawPassword, encryptedPassword);
}

// Kernel function for decryption
__global__ void decryptKernel(const char* inputEncryptedPassword, char* decryptedPassword) {
    CudaDecrypt(inputEncryptedPassword, decryptedPassword);
}

// Main function
int main() {
    // Alphabet and number set for encryption
    char cpuAlphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};
    char cpuNumbers[10] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

    // Allocate and copy alphabet and numbers to the GPU
    char* gpuAlphabet;
    char* gpuNumbers;
    hipMalloc((void**)&gpuAlphabet, sizeof(cpuAlphabet));
    hipMalloc((void**)&gpuNumbers, sizeof(cpuNumbers));
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(cpuAlphabet), hipMemcpyHostToDevice);
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(cpuNumbers), hipMemcpyHostToDevice);

    // Prompt the user to enter a raw password
    printf("Enter a Raw password (e.g., Hp45) to search for its encryption code: ");
    char rawPassword[5];
    scanf("%4s", rawPassword);

    // Allocate and copy the raw password to the GPU
    char* gpuRawPassword;
    char* gpuEncryptedPassword;
    hipMalloc((void**)&gpuRawPassword, sizeof(rawPassword));
    hipMalloc((void**)&gpuEncryptedPassword, sizeof(char) * 11);
    hipMemcpy(gpuRawPassword, rawPassword, sizeof(rawPassword), hipMemcpyHostToDevice);

    // Dynamically set the number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (26 * 26 * 10 + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the encryption kernel
    printf("\nEncrypting the provided password:\n");
    encryptKernel<<<1, 1>>>(gpuRawPassword, gpuEncryptedPassword);
    hipDeviceSynchronize();

    // Retrieve and display the encrypted password
    char encryptedPassword[11];
    hipMemcpy(encryptedPassword, gpuEncryptedPassword, sizeof(encryptedPassword), hipMemcpyDeviceToHost);
    printf("Encrypted Password: %s\n", encryptedPassword);

    // Allocate memory for decryption
    char* gpuInputEncryptedPassword;
    char* gpuDecryptedPassword;
    hipMalloc((void**)&gpuInputEncryptedPassword, sizeof(encryptedPassword));
    hipMalloc((void**)&gpuDecryptedPassword, sizeof(char) * 5);

    // Copy the encrypted password to the device for decryption
    hipMemcpy(gpuInputEncryptedPassword, encryptedPassword, sizeof(encryptedPassword), hipMemcpyHostToDevice);

    // Launch the decryption kernel
    printf("\nDecrypting the encrypted password:\n");
    decryptKernel<<<1, 1>>>(gpuInputEncryptedPassword, gpuDecryptedPassword);

    // Retrieve the decrypted password
    char decryptedPassword[5];
    hipMemcpy(decryptedPassword, gpuDecryptedPassword, sizeof(decryptedPassword), hipMemcpyDeviceToHost);

    printf("Decrypted Password: %s\n", decryptedPassword);

    // Free GPU memory
    hipFree(gpuAlphabet);
    hipFree(gpuNumbers);
    hipFree(gpuRawPassword);
    hipFree(gpuEncryptedPassword);
    hipFree(gpuInputEncryptedPassword);
    hipFree(gpuDecryptedPassword);

    return 0;
}
