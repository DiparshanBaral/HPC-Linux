#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"
#include <hip/hip_runtime.h>
#include <string.h>

// Error checking macro for CUDA functions
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Function to check if a file exists
int file_exists(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (file) {
        fclose(file);
        return 1;  // File exists
    }
    return 0;  // File does not exist
}

// Function to check if the file has a .png extension
int has_png_extension(const char *filename) {
    size_t len = strlen(filename);
    if (len > 4 && strcmp(&filename[len - 4], ".png") == 0) {
        return 1;  // Valid .png file
    }
    return 0;  // Invalid extension
}

__global__ void apply_box_blur(unsigned int h, unsigned int w, unsigned char *input_pixels, unsigned char *output_pixels) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (row * w + col) * 4;

    if (row < h && col < w) {
        int r_sum = 0, g_sum = 0, b_sum = 0, count = 0;

        // Iterate over the 3x3 box surrounding the current pixel
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int neighbor_row = row + i;
                int neighbor_col = col + j;

                // Ensure neighbors are within image bounds
                if (neighbor_row >= 0 && neighbor_row < h && neighbor_col >= 0 && neighbor_col < w) {
                    int neighbor_index = (neighbor_row * w + neighbor_col) * 4;
                    r_sum += input_pixels[neighbor_index];
                    g_sum += input_pixels[neighbor_index + 1];
                    b_sum += input_pixels[neighbor_index + 2];
                    count++;
                }
            }
        }

        // Set the new pixel value based on the average of surrounding pixels
        output_pixels[index] = r_sum / count;         // Red
        output_pixels[index + 1] = g_sum / count;    // Green
        output_pixels[index + 2] = b_sum / count;    // Blue
        output_pixels[index + 3] = input_pixels[index + 3];  // Alpha (unchanged)
    }
}

int main() {
    unsigned char *h_pixels, *h_output_pixels;
    unsigned int h, w;
    char input_filename[256], output_filename[256];

    // Ask the user for the input image file name
    do {
        printf("Enter the name of the image to be blurred (e.g., hck.png): ");
        scanf("%255s", input_filename);

        // Check if the file exists and if it's a valid PNG file
        if (!file_exists(input_filename)) {
            printf("Error: The input image file '%s' was not found. Please try again.\n", input_filename);
        } else if (!has_png_extension(input_filename)) {
            printf("Error: The input image must have a .png extension. Please try again.\n");
        }
    } while (!file_exists(input_filename) || !has_png_extension(input_filename));  // Repeat until the file exists and is a .png

    // Decode the input image
    if (lodepng_decode32_file(&h_pixels, &w, &h, input_filename) != 0) {
        fprintf(stderr, "Error: Failed to decode image %s\n", input_filename);
        return EXIT_FAILURE;
    }

    // Ask the user for the output image file name
    do {
        printf("Enter the name for the output image (e.g., blurred_image.png): ");
        scanf("%255s", output_filename);

        // Check if the output file has a valid .png extension
        if (!has_png_extension(output_filename)) {
            printf("Error: The output image must have a .png extension. Please try again.\n");
        }
    } while (!has_png_extension(output_filename));  // Repeat until a valid .png output file name is provided

    int image_size = h * w * 4;
    h_output_pixels = (unsigned char *)malloc(image_size);
    if (!h_output_pixels) {
        fprintf(stderr, "Error: Failed to allocate memory for output pixels\n");
        free(h_pixels);
        return EXIT_FAILURE;
    }

    // Allocate GPU memory
    unsigned char *d_input_pixels, *d_output_pixels;
    CUDA_CHECK(hipMalloc(&d_input_pixels, image_size));
    CUDA_CHECK(hipMalloc(&d_output_pixels, image_size));

    // Copy input image to device
    CUDA_CHECK(hipMemcpy(d_input_pixels, h_pixels, image_size, hipMemcpyHostToDevice));

    // Configure block and grid size
    dim3 blockSize(16, 16, 1);  // 16x16 threads per block
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y, 1);

    // Launch kernel to apply box blur
    apply_box_blur<<<gridSize, blockSize>>>(h, w, d_input_pixels, d_output_pixels);
    CUDA_CHECK(hipDeviceSynchronize());  // Ensure all threads complete

    // Check for any errors after kernel execution
    CUDA_CHECK(hipGetLastError());

    // Copy the blurred image back to the host
    CUDA_CHECK(hipMemcpy(h_output_pixels, d_output_pixels, image_size, hipMemcpyDeviceToHost));

    // Encode the output image and save it to file
    if (lodepng_encode32_file(output_filename, h_output_pixels, w, h) != 0) {
        fprintf(stderr, "Error: Failed to encode image %s\n", output_filename);
        free(h_pixels);
        free(h_output_pixels);
        CUDA_CHECK(hipFree(d_input_pixels));
        CUDA_CHECK(hipFree(d_output_pixels));
        return EXIT_FAILURE;
    }

    printf("Blurred image saved as %s\n", output_filename);

    // Free memory
    free(h_pixels);
    free(h_output_pixels);
    CUDA_CHECK(hipFree(d_input_pixels));
    CUDA_CHECK(hipFree(d_output_pixels));

    return 0;
}
