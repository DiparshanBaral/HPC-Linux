#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Encryption logic
__device__ char* CudaCrypt(char* rawPassword) {
    char* newPassword = (char*)malloc(sizeof(char) * 11);

    newPassword[0] = rawPassword[0] + 2;
    newPassword[1] = rawPassword[0] - 2;
    newPassword[2] = rawPassword[0] + 1;
    newPassword[3] = rawPassword[1] + 3;
    newPassword[4] = rawPassword[1] - 3;
    newPassword[5] = rawPassword[1] - 1;
    newPassword[6] = rawPassword[2] + 2;
    newPassword[7] = rawPassword[2] - 2;
    newPassword[8] = rawPassword[3] + 4;
    newPassword[9] = rawPassword[3] - 4;
    newPassword[10] = '\0';

    for (int i = 0; i < 10; i++) {
        if (i >= 0 && i < 6) { // Checking all lower case letter limits
            if (newPassword[i] > 122) {
                newPassword[i] = (newPassword[i] - 122) + 97;
            } else if (newPassword[i] < 97) {
                newPassword[i] = (97 - newPassword[i]) + 97;
            }
        } else { // Checking number section
            if (newPassword[i] > 57) {
                newPassword[i] = (newPassword[i] - 57) + 48;
            } else if (newPassword[i] < 48) {
                newPassword[i] = (48 - newPassword[i]) + 48;
            }
        }
    }
    return newPassword;
}

// Kernel function for encryption
__global__ void encryptPassword(char* inputPassword, char* encryptedPassword) {
    char* result = CudaCrypt(inputPassword);
    for (int i = 0; i < 11; i++) {
        encryptedPassword[i] = result[i];
    }
}

// Main function
int main() {
    char inputPassword[5];
    printf("Enter a 4-character password to encrypt: ");
    scanf("%4s", inputPassword);

    char* gpuInputPassword;
    char* gpuEncryptedPassword;
    char encryptedPassword[11];

    // Allocate memory on the GPU
    hipMalloc((void**)&gpuInputPassword, sizeof(char) * 5);
    hipMalloc((void**)&gpuEncryptedPassword, sizeof(char) * 11);

    // Copy input password to GPU
    hipMemcpy(gpuInputPassword, inputPassword, sizeof(char) * 5, hipMemcpyHostToDevice);

    // Launch kernel
    encryptPassword<<<1, 1>>>(gpuInputPassword, gpuEncryptedPassword);

    // Copy encrypted password back to CPU
    hipMemcpy(encryptedPassword, gpuEncryptedPassword, sizeof(char) * 11, hipMemcpyDeviceToHost);

    // Print the encrypted password
    printf("Encrypted password: %s\n", encryptedPassword);

    // Free GPU memory
    hipFree(gpuInputPassword);
    hipFree(gpuEncryptedPassword);

    return 0;
}
