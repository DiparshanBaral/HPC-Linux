#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void displayIdx(){
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    printf("idx: %d\n", idx);
}

int main(){
    dim3 gridsize(1,3,1);
    dim3 blocksize(1,2,1);
    displayIdx<<<gridsize, blocksize>>>();
    hipDeviceSynchronize();
    return 0;
}
// nvcc -o first first.cu