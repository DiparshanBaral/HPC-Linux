#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void quad(int *A, int *B, int *C, double *X1, double *X2, long lines) {
    long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < lines) { 
        double a = A[tid];
        double b = B[tid];
        double c = C[tid];
        double discriminant = b * b - 4 * a * c;

        if (discriminant >= 0) {
            X1[tid] = (-b + sqrt(discriminant)) / (2 * a);
            X2[tid] = (-b - sqrt(discriminant)) / (2 * a);
        } else { 
            X1[tid] = X2[tid] = NAN; 
        }
    }
}

int main() {
    FILE *fp = NULL;
    int a, b, c;
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    double *X1, *X2;
    double *d_X1, *d_X2;
    long lines = 0;
    long line = 0;

    // Open the file to count lines
    fp = fopen("QuadData.txt", "r");
    if (fp == NULL) {
        printf("Error opening QuadData.txt\n");
        return -1;
    }

    // Count the number of lines in the file
    while (fscanf(fp, "%d,%d,%d\n", &a, &b, &c) != EOF) {
        lines++;
    }
    fclose(fp);

    // Allocate memory on the host
    A = (int *)malloc(lines * sizeof(int));
    B = (int *)malloc(lines * sizeof(int));
    C = (int *)malloc(lines * sizeof(int));
    X1 = (double *)malloc(lines * sizeof(double));
    X2 = (double *)malloc(lines * sizeof(double));

    // Allocate memory on the device
    hipMalloc(&d_A, lines * sizeof(int));
    hipMalloc(&d_B, lines * sizeof(int));
    hipMalloc(&d_C, lines * sizeof(int));
    hipMalloc(&d_X1, lines * sizeof(double));
    hipMalloc(&d_X2, lines * sizeof(double));

    // Read the file again to fill A, B, and C
    fp = fopen("QuadData.txt", "r");
    if (fp == NULL) {
        printf("Error opening QuadData.txt\n");
        return -1;
    }

    while (fscanf(fp, "%d,%d,%d\n", &a, &b, &c) != EOF) {
        A[line] = a;
        B[line] = b;
        C[line] = c;
        line++;
    }
    fclose(fp);

    // Copy data from host to device
    hipMemcpy(d_A, A, lines * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, lines * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, lines * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (lines + threadsPerBlock - 1) / threadsPerBlock;
    quad<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_X1, d_X2, lines);

    // Copy results back to host
    hipMemcpy(X1, d_X1, lines * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(X2, d_X2, lines * sizeof(double), hipMemcpyDeviceToHost);

    // Print results
    for (long i = 0; i < lines; i++) {
        if (!isnan(X1[i]) && !isnan(X2[i])) {
            printf("Row %ld: X1 = %lf, X2 = %lf\n", i, X1[i], X2[i]);
        } else {
            printf("Row %ld: Roots are complex or undefined\n", i);
        }
    }

    // Free memory
    free(A);
    free(B);
    free(C);
    free(X1);
    free(X2);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X1);
    hipFree(d_X2);

    return 0;
}