#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void display() {
    int threadID = threadIdx.x;
    int blockID = blockIdx.x;
    printf("Block %d -> ThreadId = %d\n", blockID, threadID);
}

int main() {
    dim3 gridSize(1, 1, 1);   
    dim3 blockSize(5, 1, 1);  
    display<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}
