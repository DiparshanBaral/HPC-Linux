#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void display() {
    int threadIDX = threadIdx.x;
    int threadIDY = threadIdx.y;
    int threadIDZ = threadIdx.z;
    int blockID = blockIdx.x;
    printf("Block %d -> ThreadId = %d,ThreadID = %d, ThreadId = %d\n ", blockID, threadIDX, threadIDY, threadIDZ);
}

int main() {
    dim3 gridSize(1, 1, 1);   
    dim3 blockSize(5, 2, 3);  
    display<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}
