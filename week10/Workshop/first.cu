//Does the code work? If not, what are the errors?


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void displayHelloWorld(){
    printf("Hello World\n");
}

__global__ void display(){
    displayHelloWorld();
}

int main(){
    display<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
