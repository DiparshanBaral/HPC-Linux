#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void displayIdx()
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    printf("%d\n", tid);
}

int main()
{
    int n;
    printf("Enter n : ");
    scanf("%d", &n);
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(n, n, n);
    displayIdx<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}
