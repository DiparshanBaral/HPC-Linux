#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


// Function to compare two strings on the device
__device__ bool compareStrings(char *stringA, char *stringB)
{
    int index = 0;
    while (stringA[index] == stringB[index])
    {
        if (stringA[index] == '\0' || stringB[index] == '\0')
            break;
        index++;
    }
    return (stringA[index] == '\0' && stringB[index] == '\0');
}

// Device function to encrypt the password
__device__ void CudaCrypt(char *rawPassword, char *newPassword)
{
    newPassword[0] = rawPassword[0] + 2;
    newPassword[1] = rawPassword[0] - 2;
    newPassword[2] = rawPassword[0] + 1;
    newPassword[3] = rawPassword[1] + 3;
    newPassword[4] = rawPassword[1] - 3;
    newPassword[5] = rawPassword[1] - 1;
    newPassword[6] = rawPassword[2] + 2;
    newPassword[7] = rawPassword[2] - 2;
    newPassword[8] = rawPassword[3] + 4;
    newPassword[9] = rawPassword[3] - 4;
    newPassword[10] = '\0';

    for (int i = 0; i < 10; i++)
    {
        if (i < 6)
        {
            if (newPassword[i] > 122)
                newPassword[i] = (newPassword[i] - 122) + 97;
            else if (newPassword[i] < 97)
                newPassword[i] = (97 - newPassword[i]) + 97;
        }
        else
        {
            if (newPassword[i] > 57)
                newPassword[i] = (newPassword[i] - 57) + 48;
            else if (newPassword[i] < 48)
                newPassword[i] = (48 - newPassword[i]) + 48;
        }
    }
}

// Kernel function to decrypt the password
__global__ void decrypt(char *alphabet, char *numbers, char *encPass, char *outputDevice)
{
    char genRawPass[4];
    genRawPass[0] = alphabet[blockIdx.x];
    genRawPass[1] = alphabet[blockIdx.y];
    genRawPass[2] = numbers[threadIdx.x];
    genRawPass[3] = numbers[threadIdx.y];

    char enc[11]; // Space to hold encrypted password
    CudaCrypt(genRawPass, enc);

    if (compareStrings(enc, encPass))
    {
        printf("Password found: %s\n", genRawPass);
        for (int i = 0; i < 4; i++)
            outputDevice[i] = genRawPass[i];
    }
}

int main(int argc, char **argv)
{
    char *outputHost = (char *)malloc(sizeof(char) * 4);
    char *outputDevice;

    hipMalloc((void **)&outputDevice, sizeof(char) * 4);

    // Input encrypted password
    char inputHost[11];
    printf("Enter the encrypted password: ");
    scanf("%s", inputHost);
    inputHost[10] = '\0';

    char *inputDevice;
    hipMalloc((void **)&inputDevice, sizeof(char) * 11);
    hipMemcpy(inputDevice, inputHost, sizeof(char) * 11, hipMemcpyHostToDevice);

    // Alphabet and number arrays
    char cpuAlphabet[26] = {'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'};
    char cpuNumbers[10] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

    char *gpuAlphabet, *gpuNumbers;
    hipMalloc((void **)&gpuAlphabet, sizeof(char) * 26);
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    hipMalloc((void **)&gpuNumbers, sizeof(char) * 10);
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

    // Launch kernel with multiple blocks and threads
    dim3 grid(26, 26, 1);
    dim3 block(10, 10, 1);
    decrypt<<<grid, block>>>(gpuAlphabet, gpuNumbers, inputDevice, outputDevice);
    hipDeviceSynchronize();

    // Copy decrypted password back to host
    hipMemcpy(outputHost, outputDevice, sizeof(char) * 4, hipMemcpyDeviceToHost);

    printf("The final password is: %s\n", outputHost);

    // Free memory
    hipFree(outputDevice);
    hipFree(inputDevice);
    hipFree(gpuAlphabet);
    hipFree(gpuNumbers);
    free(outputHost);

    return 0;
}