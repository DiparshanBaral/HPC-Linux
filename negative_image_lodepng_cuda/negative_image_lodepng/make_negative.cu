#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"
#include <hip/hip_runtime.h>

__global__ void manipulate_pixels(unsigned int h, unsigned int w, unsigned char *pixels) {
	int rows = blockIdx.y * blockDim.y + threadIdx.y;
	int cols = blockIdx.x * blockDim.x + threadIdx.x;
        int index = (rows * w + cols) * 4;
//	printf("index = %d, pixel[%d]= %d\n", index, index, pixels[index]);
	if(rows<h && cols<w){
            		pixels[index] = 255 - pixels[index];		//Red
			pixels[index + 1] = 255 - pixels[index+1];	//Green
            		pixels[index + 2] = 255 - pixels[index+2];	//Blue
            		pixels[index + 3] = 255;          // Alpha (fully opaque)
        }
}

int main() {
	unsigned char *h_pixels;
	unsigned int h, w;

	lodepng_decode32_file(&h_pixels, &w, &h,"eagle.png");

	unsigned char *d_pixels;
	int image_size = h*w*4;
	hipMalloc(&d_pixels,image_size);
	hipMemcpy(d_pixels, h_pixels, image_size, hipMemcpyHostToDevice);

	dim3 gridSize(w,h,1);
	dim3 blockSize(4,1,1);

	manipulate_pixels<<<gridSize,blockSize>>>(h,w,d_pixels);
	hipDeviceSynchronize();

	hipMemcpy(h_pixels, d_pixels, image_size, hipMemcpyDeviceToHost);

	lodepng_encode32_file("negative.png",h_pixels,w,h);

    	free(h_pixels);
	hipFree(d_pixels);
    	return 0;
}
