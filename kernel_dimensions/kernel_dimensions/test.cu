
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void display(){
      int i = 
    threadIdx.x +
    (threadIdx.y * blockDim.x) +
    (threadIdx.z * blockDim.x * blockDim.y) + 
    (blockIdx.x * blockDim.x * blockDim.y * blockDim.z) +
    (blockIdx.y * blockDim.x * blockDim.y * blockDim.z * gridDim.x) +
    (blockIdx.z * blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y);
    printf("tdx: %d\n", i);

}

int main(){
    dim3 gridDim(13,13, 2);
    dim3 blockDim(10,10, 2);
    display<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    return 0;
}